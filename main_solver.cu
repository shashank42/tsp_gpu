#include "hip/hip_runtime.h"
/*
    columbus: Software for computing approximate solutions to the traveling
   salesman's problem on GPUs Copyright (C) 2016 Steve Bronder and Haoyan Min

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as published
    by the Free Software Foundation, either version 3 of the License, or
    any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details.

    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <assert.h>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <fstream>
#include <iostream>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
// If NDEBUG is defined, cudaCheckError() will be empty
#define NDEBUG
#include "kernels/utils.h"
#include "kernels/initialize_rng.h"
#include "kernels/insert_sampler.h"
#include "kernels/opt2_sampler.h"
#include "kernels/swap_sampler.h"


#define t_num 1024
#define GRID_SIZE 131072

/*
For more samples define GRID_SIZE as a multiple of t_num such as 512000,
2048000, or the (max - 1024) grid size 2147482623 A good grid size is the number
of SM's you have times the number of blocks each can take in times max threads
per block I have 8 cores that can hold 16 blocks of 1024 cores so my best is
131072 Some compiler options that can speed things up
--use_fast_math
--optimize=5
--gpu-architecture=compute_35
I use something like
NOTE: You need to use the -lcurand flag to compile for the RNG.
nvcc --optimize=5 --use_fast_math -arch=compute_35 main_solver.cu -o tsp_cuda
-lcurand
*/

int main(int argc, char *argv[]) {

  // Reading in inputs
  if (argc == 1) {
    printf(
        "Inputs: \n"
        "(Required) input_file.tsp: [char()] \n"
        " - The name of the tsp file, excluding .tsp at the end, containing "
        "the cities to travel over. \n"
        "(Optional) -trip: [char()] \n"
        " - The name of the csv file, excluding .csv, containing a previously "
        "found trip."
        " If missing, a linear route is generated as the starting trip. \n"
        "(Optional) -temp: [float(1)] \n"
        " - The initial starting temperature. Default is 1000 \n"
        "(Optional) -decay: [float(1)]  \n"
        " - The decay rate for the annealing schedule. Default is .99 \n"
        "(Optional) -maxiter: [integer(1)]  \n"
        " - The maximum number of iterations until failure. \n"
        "  Default is -1, which runs until temperature goes to the minimum.\n"
        "(Optional) -global_search: [float(1)]  \n"
        " - A parameter that controls the variance of the second city search "
        "space,\n"
        "   such that the variance is [30 + exp(global_search/Temp) * N]. "
        "default is .01.\n"
        "  See An example of what this controls here:\n"
        "(Optional) -local_search: [float(1)]  \n"
        " - A parameter that controls the variance of the second city search "
        "space,\n"
        "   such that the variance is [30 + exp(local_search/Temp) * N]. "
        "default is 1.\n");

    return 1;
  }

  const char *tsp_name = concat(argv[1], ".tsp");
  coordinates *location_g;
  read_tsp(tsp_name);
  unsigned int N = meta->dim, *N_g;
  unsigned int i;
  unsigned int *salesman_route =
      (unsigned int *)malloc((N + 1) * sizeof(unsigned int));
  float sample_area_local, sample_area_global, *sample_area_local_g,
      *sample_area_global_g;
  sample_area_global = 0.01;
  sample_area_local = 1;
  // just make one inital guess route, a simple linear path
  for (i = 0; i <= N; i++)
    salesman_route[i] = i;

  // Set the starting and end points to be the same
  salesman_route[N] = salesman_route[0];

  // Get loss
  float T[1], *T_g;
  T[1] = .03;
  float decay = 0.99;
  int maxiter = -1;
  // Get starting trip
  for (i = 0; i <= N; i++)
    salesman_route[i] = i;
  // Set the starting and end points to be the same
  salesman_route[N] = salesman_route[0];

  // read in options
  for (int i = 1; i < argc; i++) {
    if (i + 1 != argc) {
      if (strcmp(argv[i], "-trip=") == 0) {
        const char *trip_name = concat(argv[i + 1], ".csv");
        read_trip(trip_name, salesman_route);
      }
      if (strcmp(argv[i], "-temp=") == 0) {
        // If atof cannot convert to a float, it returns 0
        float user_temp = atof(argv[i + 1]);
        if (user_temp == 0) {
          printf("Error: Initial Temperature must be a non-zero number\n");
          return 1;
        }
        T[0] = user_temp;
        T[1] = T[0];
      }
      if (strcmp(argv[i], "-maxiter=") == 0) {
        // If atof cannot convert to a float, it returns 0
        float user_iter = atoi(argv[i + 1]);
        if (user_iter == 0) {
          printf("Error: max iter cannot be zero\n");
          return 1;
        }
        maxiter = user_iter;
      }
      if (strcmp(argv[i], "-decay=") == 0) {
        // If atoi cannot convert to number, it returns 0
        float user_decay = atof(argv[i + 1]);
        if (user_decay == 0) {
          printf("Error: Decay must be a number from 0 to 1\n");
          return 1;
        } else if (user_decay >= 1 || user_decay <= 0) {
          printf("Error: Decay must be a number from 0 to 1\n");
          return 1;
        } else {
          decay = user_decay;
        }
      }
      if (strcmp(argv[i], "-global_search=") == 0) {
        // If atoi cannot convert to number, it returns 0
        float user_global = atof(argv[i + 1]);
        if (user_global == 0) {
          printf("Error: global search param must be greater than 0. \n");
          return 1;
        } else {
          sample_area_global = user_global;
        }
      }
      if (strcmp(argv[i], "-local_search=") == 0) {
        // If atoi cannot convert to number, it returns 0
        float user_local = atof(argv[i + 1]);
        if (user_local == 0) {
          printf("Error: local search param must be greater than 0. \n");
          return 1;
        } else {
          sample_area_local = user_local;
        }
      }
    }
  }

  // Calculate the original loss
  float original_loss = 0;
  for (i = 0; i < N; i++) {
    original_loss += sqrtf(
        (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
            (location[salesman_route[i]].x -
             location[salesman_route[i + 1]].x) +
        (location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
            (location[salesman_route[i]].y -
             location[salesman_route[i + 1]].y));
  }
  printf("Original Loss is:  %0.6f \n", original_loss);
  float optimized_loss_restart = original_loss;
  // Keep the original loss for comparison pre/post algorithm
  // SET THE LOSS HERE

  /*
  Defining device variables:
  city_swap_one_h/g: [integer(t_num)]
  - Host/Device memory for city one
  city_swap_two_h/g: [integer(t_num)]
  - Host/Device memory for city two
  flag_h/g: [integer(t_num)]
  - Host/Device memory for flag of accepted step
  salesman_route_g: [integer(N)]
  - Device memory for the salesmans route
  flag_h/g: [integer(t_num)]
  - host/device memory for acceptance vector
  original_loss_g: [integer(1)]
  - The device memory for the current loss function
  (DEPRECATED)new_loss_h/g: [integer(t_num)]
  - The host/device memory for the proposal loss function
  */
  unsigned int *city_swap_one_h =
      (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
  unsigned int *city_swap_two_h =
      (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
  unsigned int *flag_h =
      (unsigned int *)malloc(GRID_SIZE * sizeof(unsigned int));
  unsigned int *salesman_route_g, *salesman_route_2g, *salesman_route_restartg,
      *flag_g, *city_swap_one_g, *city_swap_two_g;
  int global_flag_h = -1, *global_flag_g;

  hipMalloc((void **)&city_swap_one_g, GRID_SIZE * sizeof(unsigned int));
  cudaCheckError();
  hipMalloc((void **)&city_swap_two_g, GRID_SIZE * sizeof(unsigned int));
  cudaCheckError();
  hipMalloc((void **)&location_g, N * sizeof(coordinates));
  cudaCheckError();
  hipMalloc((void **)&salesman_route_g, (N + 1) * sizeof(unsigned int));
  cudaCheckError();
  hipMalloc((void **)&salesman_route_2g, (N + 1) * sizeof(unsigned int));
  cudaCheckError();
  hipMalloc((void **)&salesman_route_restartg, (N + 1) * sizeof(unsigned int));
  cudaCheckError();
  hipMalloc((void **)&T_g, sizeof(float));
  cudaCheckError();
  hipMalloc((void **)&sample_area_global_g, sizeof(float));
  cudaCheckError();
  hipMalloc((void **)&sample_area_local_g, sizeof(float));
  cudaCheckError();
  hipMalloc((void **)&flag_g, GRID_SIZE * sizeof(int));
  cudaCheckError();
  hipMalloc((void **)&global_flag_g, sizeof(int));
  cudaCheckError();
  hipMalloc((void **)&N_g, sizeof(unsigned int));
  cudaCheckError();

  hipMemcpy(location_g, location, N * sizeof(coordinates),
             hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(salesman_route_g, salesman_route, (N + 1) * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(salesman_route_2g, salesman_route, (N + 1) * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(salesman_route_restartg, salesman_route,
             (N + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
             hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(N_g, &N, sizeof(unsigned int), hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(sample_area_global_g, &sample_area_global, sizeof(unsigned int),
             hipMemcpyHostToDevice);
  cudaCheckError();
  hipMemcpy(sample_area_local_g, &sample_area_local, sizeof(unsigned int),
             hipMemcpyHostToDevice);
  cudaCheckError();
  // Beta is the decay rate
  // float beta = 0.0001;
  // We are going to try some stuff for temp from this adaptive simulated
  // annealing paper https://arxiv.org/pdf/cs/0001018.pdf

  // Number of thread blocks in grid
  // X is for the sampling, y is for manipulating the salesman's route
  dim3 blocksPerSampleGrid(GRID_SIZE / t_num, 1, 1);
  dim3 blocksPerTripGrid((N / t_num) + 1, 1, 1);
  dim3 threadsPerBlock(t_num, 1, 1);

  // Trying out random gen in cuda
  hiprandState_t *states;

  /* allocate space on the GPU for the random states */
  hipMalloc((void **)&states, GRID_SIZE * sizeof(hiprandState_t));
  init<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(time(0), states);

  // time counter
  time_t t_start, t_end;
  t_start = time(NULL);
  long int iter = 1;
  printf("Ending Temp: %f \n", FLT_EPSILON * 100);
  // int sames = 0;
  printf(" Loss | Temp | Iter | Time \n");
  while (T[0] > FLT_EPSILON * 100 | T[0] == 0.0) {
    // Copy memory from host to device
    hipMemcpy(T_g, T, sizeof(float), hipMemcpyHostToDevice);
    i = 1;

    while (i < 2000) { // key

      // two opt
      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();

      twoOptStep<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, location_g, salesman_route_g, T_g,
          global_flag_g, N_g, states, sample_area_global_g);
      cudaCheckError();

      opt2Update<<<blocksPerTripGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();

      twoOptStep<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, location_g, salesman_route_g, T_g,
          global_flag_g, N_g, states, sample_area_local_g);
      cudaCheckError();

      opt2Update<<<blocksPerTripGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      // insertionstep
      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();

      insertionStep<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, location_g, salesman_route_g, T_g,
          global_flag_g, N_g, states, sample_area_global_g);
      cudaCheckError();

      insertionUpdate<<<blocksPerTripGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      insertionUpdateEndPoints<<<blocksPerTripGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();

      insertionStep<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, location_g, salesman_route_g, T_g,
          global_flag_g, N_g, states, sample_area_local_g);
      cudaCheckError();

      insertionUpdate<<<blocksPerTripGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      insertionUpdateEndPoints<<<blocksPerTripGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();
      // swap step

      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      swapStep<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, location_g, salesman_route_g, T_g,
          global_flag_g, N_g, states, sample_area_global_g);
      cudaCheckError();

      swapUpdate<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();

      swapStep<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, location_g, salesman_route_g, T_g,
          global_flag_g, N_g, states, sample_area_local_g);
      cudaCheckError();

      swapUpdate<<<blocksPerSampleGrid, threadsPerBlock, 0>>>(
          city_swap_one_g, city_swap_two_g, salesman_route_g, salesman_route_2g,
          global_flag_g);
      cudaCheckError();

      hipMemcpy(global_flag_g, &global_flag_h, sizeof(int),
                 hipMemcpyHostToDevice);
      cudaCheckError();

      hipMemcpy(salesman_route_2g, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();

      i++;
    }
    hipMemcpy(salesman_route, salesman_route_g, (N + 1) * sizeof(unsigned int),
               hipMemcpyDeviceToHost);
    cudaCheckError();
    float optimized_loss = 0;
    for (i = 0; i < N; i++) {
      optimized_loss += sqrt(
          (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
              (location[salesman_route[i]].x -
               location[salesman_route[i + 1]].x) +
          (location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
              (location[salesman_route[i]].y -
               location[salesman_route[i + 1]].y));
    }
    printf(" %.6f | %f | %ld | %f\n", optimized_loss, T[0], iter,
           difftime(time(NULL), t_start));
    T[0] = T[0] * decay;
    iter++;
    // This grabs the best trip overall
    if (optimized_loss < optimized_loss_restart) {
      optimized_loss_restart = optimized_loss;
      hipMemcpy(salesman_route_restartg, salesman_route_g,
                 (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToDevice);
      cudaCheckError();
      // sames = 0;
    } /*else if (abs(optimized_loss - optimized_loss_restart) < 2){
    // If we are only gaining by one then we can start speeding things up
        sames++;
        if (sames > 10){
            T[0] = T[0] * 0.8;
            }
    }*/
    if (maxiter > 0 && maxiter < iter)
      break;
  }

  t_end = time(NULL);
  printf("time = %f\n", difftime(t_end, t_start));

  hipMemcpy(salesman_route, salesman_route_g, (N + 1) * sizeof(unsigned int),
             hipMemcpyDeviceToHost);
  cudaCheckError();

  // We have to redefine optimized loss for some reason?
  float optimized_loss = 0;
  for (i = 0; i < N; i++) {
    optimized_loss += sqrt(
        (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
            (location[salesman_route[i]].x -
             location[salesman_route[i + 1]].x) +
        (location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
            (location[salesman_route[i]].y -
             location[salesman_route[i + 1]].y));
  }

  // If it's worse than the restart make the route the restart.
  if (optimized_loss > optimized_loss_restart) {
    hipMemcpy(salesman_route, salesman_route_restartg,
               (N + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
    cudaCheckError();
  }

  optimized_loss = 0;
  for (i = 0; i < N; i++) {
    optimized_loss += sqrt(
        (location[salesman_route[i]].x - location[salesman_route[i + 1]].x) *
            (location[salesman_route[i]].x -
             location[salesman_route[i + 1]].x) +
        (location[salesman_route[i]].y - location[salesman_route[i + 1]].y) *
            (location[salesman_route[i]].y -
             location[salesman_route[i + 1]].y));
  }

  printf("Original Loss is:  %0.6f \n", original_loss);
  printf("Optimized Loss is: %.6f \n", optimized_loss);

  // Write the best trip to CSV
  FILE *best_trip;
  const char *filename = concat(argv[1], "_trip.csv");
  best_trip = fopen(filename, "w+");
  fprintf(best_trip, "location,coordinate_x,coordinate_y\n");
  for (i = 0; i < N + 1; i++) {
    fprintf(best_trip, "%d,%.6f,%.6f\n", salesman_route[i],
            location[salesman_route[i]].x, location[salesman_route[i]].y);
  }
  fclose(best_trip);

  hipFree(location_g);
  cudaCheckError();
  hipFree(salesman_route_g);
  cudaCheckError();
  hipFree(salesman_route_2g);
  cudaCheckError();
  hipFree(T_g);
  cudaCheckError();
  hipFree(flag_g);
  cudaCheckError();
  hipFree(global_flag_g);
  cudaCheckError();
  hipFree(salesman_route_restartg);
  cudaCheckError();
  hipFree(sample_area_global_g);
  cudaCheckError();
  hipFree(sample_area_local_g);
  cudaCheckError();
  free(salesman_route);
  free(city_swap_one_h);
  free(city_swap_two_h);
  free(flag_h);
  free(location);
  return 0;
}
